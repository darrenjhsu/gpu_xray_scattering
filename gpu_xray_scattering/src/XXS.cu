
#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "kernel.cu"
#include "WaasKirf.hh"
#include "vdW.hh"
#include "XS.hh"
#define PI 3.14159265359 


extern "C" {
void cross_xray_scattering (
    int num_atom1,    // number of atoms Na
    float *coord1,    // coordinates     3Na
    int num_atom2,
    float *coord2,
    int *Ele1,      // Element
    int *Ele2,
    float *weight,   // Weights of the protein and prior points
    int num_q,       // number of q vector points
    float *q,        // q vector        Nq
    float *S_calc1,   // scattering intensity to be returned, Nq
    float *S_calc2,
    float *S_calc12,
    int num_q_raster // number of q_raster points when using scat_calc_oa
) {
// This function is called by tclforce script from NAMD. Note that it is only executed every delta_t steps.
    struct timeval tv1, tv2;
    gettimeofday(&tv1, NULL);

    // In this code pointers with d_ are device pointers. 
    int num_atom1_1024 = (num_atom1 + 1023) / 1024 * 1024;
    int num_atom2_1024 = (num_atom2 + 1023) / 1024 * 1024;
    int num_q2        = (num_q + 31) / 32 * 32;
    int num_q_raster2 = (num_q_raster + 1023) / 1024 * 1024;


    // Declare cuda pointers //
    float *d_coord1;          // Coordinates 3 x num_atom
    float *d_coord2;
    int   *d_Ele1;            // Element list.
    int   *d_Ele2;
    float *d_weight;
    float *d_q;              // q vector
    float *d_S_calc1;         // Calculated scattering curve
    float *d_S_calcc1;        // Some intermediate matrices
    float *d_S_calc2;         // Calculated scattering curve
    float *d_S_calcc2;        // Some intermediate matrices
    float *d_S_calc12;         // Calculated scattering curve
    float *d_S_calcc12;        // Some intermediate matrices

    float *d_WK;             // Waasmaier-Kirfel parameters 

    float *d_FF_table,       // Form factors for each atom type at each q
          *d_FF_full1,        /* Form factors for each atom at each q, 
                                considering the SASA an atom has. */
          *d_FF_full2;
    
    // set various memory chunk sizes
    int size_coord1      = 3 * num_atom1 * sizeof(float);
    int size_coord2      = 3 * num_atom2 * sizeof(float);
    int size_atom1       = num_atom1 * sizeof(int);
    int size_atom1f       = num_atom1 * sizeof(float);
    int size_atom2       = num_atom2 * sizeof(int);
    int size_q           = num_q * sizeof(float); 
    int size_qxatom2     = num_q2 * 1024 * sizeof(float);
    int size_qxqraster2  = num_q2 * num_q_raster2 * sizeof(float);
    int size_FF_table    = (num_ele) * num_q * sizeof(float);
    int size_FF_full1    = num_q * num_atom1_1024 * sizeof(float);
    int size_FF_full2    = num_q * num_atom2_1024 * sizeof(float);
    int size_WK          = 11 * num_ele * sizeof(float);


    // Allocate cuda memories
    hipMalloc((void **)&d_coord1,      size_coord1); // 40 KB
    hipMalloc((void **)&d_coord2,      size_coord2); // 40 KB
    hipMalloc((void **)&d_Ele1,        size_atom1);
    hipMalloc((void **)&d_Ele2,        size_atom2);
    hipMalloc((void **)&d_weight,      size_atom1f);
    hipMalloc((void **)&d_q,           size_q);
    hipMalloc((void **)&d_S_calc1,     size_q);
    hipMalloc((void **)&d_S_calc2,     size_q);
    hipMalloc((void **)&d_S_calc12,    size_q);
    hipMalloc((void **)&d_S_calcc1,    size_qxqraster2);
    hipMalloc((void **)&d_S_calcc2,    size_qxqraster2);
    hipMalloc((void **)&d_S_calcc12,   size_qxqraster2);
    hipMalloc((void **)&d_FF_table,    size_FF_table);
    hipMalloc((void **)&d_FF_full1,    size_FF_full1);
    hipMalloc((void **)&d_FF_full1,    size_FF_full2);
    hipMalloc((void **)&d_WK,          size_WK);

    // Initialize some matrices
    
    hipMemset(d_FF_full1,    0.0, size_FF_full1);
    hipMemset(d_FF_full2,    0.0, size_FF_full2);

    hipMemset(d_S_calc1,     0.0, size_q);
    hipMemset(d_S_calc2,     0.0, size_q);
    hipMemset(d_S_calc12,    0.0, size_q);
    hipMemset(d_S_calcc1,    0.0, size_qxqraster2);
    hipMemset(d_S_calcc2,    0.0, size_qxqraster2);
    hipMemset(d_S_calcc12,   0.0, size_qxqraster2);

    // Copy necessary data
    hipMemcpy(d_coord1,     coord1,     size_coord1, hipMemcpyHostToDevice);
    hipMemcpy(d_coord2,     coord2,     size_coord2, hipMemcpyHostToDevice);
    hipMemcpy(d_Ele1,       Ele1,       size_atom1,  hipMemcpyHostToDevice);
    hipMemcpy(d_Ele2,       Ele2,       size_atom2,  hipMemcpyHostToDevice);
    hipMemcpy(d_weight,     weight,     size_atom1f, hipMemcpyHostToDevice);
    hipMemcpy(d_q,          q,          size_q,      hipMemcpyHostToDevice);
    hipMemcpy(d_WK,         WK,         size_WK,     hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: setting memory, %s\n", hipGetErrorString(error) );
       exit(-1);
    }

    // Register what atoms are close to what atoms. 

    // Calculate the non-varying part of the form factor.
    // In the future this can be done pre-simulation.
    FF_calc<<<num_q, 32>>>(
        d_q, 
        d_WK, 
        num_q, 
        num_ele, 
        d_FF_table
        );

    // Adding the surface area contribution. From this point every atom has a different form factor.
    create_FF_full_FoXS<<<num_q, 1024>>>(
        d_FF_table1, 
        d_Ele1, 
        d_FF_full, 
        num_q, 
        num_ele1, 
        num_atom1, 
        num_atom1_1024);

    create_FF_full_FoXS<<<num_q, 1024>>>(
        d_FF_table2,
        d_Ele2,
        d_FF_full, 
        num_q,
        num_ele2, 
        num_atom2, 
        num_atom2_1024);

    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: FF %s\n", hipGetErrorString(error) );
       exit(-1);
    }

    // Actually calculating scattering pattern. This kernel is for single snapshot  
    printf("Using orientational averaging method 1\n");
    scat_calc_xoa<<<num_q, 1024>>>(
        d_coord1, 
        d_coord2,
        d_Ele1,
        d_Ele2,
        d_weight,
        d_q, 
        d_S_calc1, 
        d_S_calc2, 
        d_S_calc12, 
        num_atom1,
        num_atom2,
        num_q,     
        num_ele1,
        num_ele2,
        d_S_calcc1, 
        d_S_calcc2, 
        d_S_calcc12, 
        num_atom1_1024,
        num_atom2_1024,
        d_FF_full1,
        d_FF_full2,
        num_q_raster,
        num_q_raster2);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: scat_calc %s\n", hipGetErrorString(error) );
       exit(-1);
    }

    hipMemcpy(S_calc1, d_S_calc1, size_q,     hipMemcpyDeviceToHost);
    hipMemcpy(S_calc2, d_S_calc2, size_q,     hipMemcpyDeviceToHost);
    hipMemcpy(S_calc12, d_S_calc12, size_q,     hipMemcpyDeviceToHost);
   

    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
       exit(-1);
    }

    hipFree(d_coord1); 
    hipFree(d_coord2); 
    hipFree(d_Ele1); 
    hipFree(d_Ele2); 
    hipFree(d_weight); 
    hipFree(d_q);
    hipFree(d_S_calc1); 
    hipFree(d_S_calc2); 
    hipFree(d_S_calc12); 
    hipFree(d_S_calcc1); 
    hipFree(d_S_calcc2); 
    hipFree(d_S_calcc12); 
    hipFree(d_WK);
    hipFree(d_FF_table); 
    hipFree(d_FF_full1);
    hipFree(d_FF_full2);
    gettimeofday(&tv2, NULL);
    double time_in_mill = 
         (tv2.tv_sec - tv1.tv_sec) * 1000.0 + (tv2.tv_usec - tv1.tv_usec) / 1000.0 ;

}
}
