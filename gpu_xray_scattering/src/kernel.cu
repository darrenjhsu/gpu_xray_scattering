#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <time.h>
#include "WaasKirf.hh"
#define PI 3.14159265359


__global__ void dist_calc (
    float *coord, 
    int *close_num,
    int *close_flag, 
    int *close_idx, 
    int num_atom, 
    int num_atom2) {

    // close_flag is a 1024 x num_atom2 int matrix initialized to 0.
    // close_idx: A 128 x num_atom2 int matrix, row i of which only the first close_num[i] elements are defined. (Otherwise it's -1). 
    __shared__ float x_ref, y_ref, z_ref;
    __shared__ int idz;
    __shared__ int temp[2048];

    __shared__ int round;
    round = (num_atom2 > num_atom) ? 1 : (num_atom + num_atom2 - 1) / num_atom2;
    /*if (blockIdx.x == 0) {
        if (threadIdx.x == 0) printf("Number of rounds: %d\n", round);
    }*/
    // Calc distance
    for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
        if (threadIdx.x == 0) {
            x_ref = coord[3*ii  ];
            y_ref = coord[3*ii+1];
            z_ref = coord[3*ii+2];
        }
        int idy = ii % gridDim.x; // This will be what row of close_flag this block is putting its value in.
        __syncthreads();

        for (int rr = 0; rr < round; rr++) {
            for (int jj = threadIdx.x; jj < num_atom2; jj += blockDim.x) { // flush close_flag
                close_flag[idy*num_atom2+jj] = 0;
            }

            for (int jj = threadIdx.x + rr * num_atom2; jj < num_atom; jj += blockDim.x) {
                float r2t = (coord[3*jj  ] - x_ref) * (coord[3*jj  ] - x_ref) + 
                            (coord[3*jj+1] - y_ref) * (coord[3*jj+1] - y_ref) + 
                            (coord[3*jj+2] - z_ref) * (coord[3*jj+2] - z_ref); 
     
                if (r2t < 34.0) {
                    close_flag[idy*num_atom2+jj % num_atom2] = 1; // roughly 2 A + 2 A vdW + 2 * 1.8 A probe
                } else { 
                    close_flag[idy*num_atom2+jj % num_atom2] = 0;
                }
                if (ii == jj) close_flag[idy*num_atom2+jj % num_atom2] = 0;
            }
            __syncthreads();
    
            // Prefix sum: Do pre scan
            idz = 0;
            int temp_sum = 0;
            for (int jj = threadIdx.x; jj < num_atom2; jj += 2 * blockDim.x) {
                int idx = jj % blockDim.x; 
                int offset = 1;
                temp[2 * idx]     = close_flag[idy * num_atom2 + 2 * blockDim.x * idz + 2 * idx];
                temp[2 * idx + 1] = close_flag[idy * num_atom2 + 2 * blockDim.x * idz + 2 * idx + 1];
                for (int d = 2 * blockDim.x>>1; d > 0; d >>= 1) { // up-sweep
                    __syncthreads();
                    if (idx < d) {
                        int ai = offset * (2 * idx + 1) - 1;
                        int bi = offset * (2 * idx + 2) - 1;
                        temp[bi] += temp[ai];
                    }
                    offset *= 2;
                }
                __syncthreads();
                temp_sum = close_num[ii];
                __syncthreads();
                if (idx == 0) {
                    close_num[ii] += temp[2 * blockDim.x - 1]; // log the total number of 1's in this blockDim
                    temp[2 * blockDim.x - 1] = 0;
                }
                __syncthreads();
                for (int d = 1; d < blockDim.x * 2; d *= 2) { //down-sweep
                    offset >>= 1;
                    __syncthreads();
                    if (idx < d) {
                        int ai = offset * (2 * idx + 1) - 1;
                        int bi = offset * (2 * idx + 2) - 1;
                        int t    = temp[ai];
                        temp[ai] = temp[bi];
                        temp[bi] += t;
                    }
                }
            
                __syncthreads();
            
                // Finally assign the indices
                if (close_flag[idy * num_atom2 + 2 * blockDim.x * idz + 2 * idx] == 1) {
                    close_idx[ii * 128 + temp[2*idx] + temp_sum] = 2 * idx + 2 * blockDim.x * idz;
                }
                if (close_flag[idy * num_atom2 + 2 * blockDim.x * idz + 2 * idx + 1] == 1) {
                    close_idx[ii * 128 + temp[2*idx+1] + temp_sum] = 2 * idx + 1 + 2 * blockDim.x * idz;
                }
                idz++;
                __syncthreads();
            } // prefix sum loop
        } // round loop
    } // block loop
} // function


__global__ void __launch_bounds__(512,4) surf_calc (
    float *coord, 
    int *Ele, 
    int *close_num, 
    int *close_idx, 
    float *vdW, 
    int num_atom, 
    int num_atom2, 
    int num_raster, 
    float sol_s, 
    float *V) {

    // num_raster should be a number of 2^n. 
    // sol_s is solvent radius (default = 1.8 A)
    __shared__ float vdW_s; // vdW radius of the center atom
    __shared__ int pts[512]; // All spherical raster points
    __shared__ float L, r;
    
    if (blockIdx.x >= num_atom) return;
    L = sqrt(num_raster * PI);
    for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
        int atom1t = Ele[ii];
        vdW_s = vdW[atom1t];
        r = vdW_s + sol_s;
        for (int jj = threadIdx.x; jj < num_raster; jj += blockDim.x) {
            int pt = 1;
            
            float h = 1.0 - (2.0 * (float)jj + 1.0) / (float)num_raster;
            float p = acos(h);
            float t = L * p; 
            float xu = sin(p) * cos(t);
            float yu = sin(p) * sin(t);
            float zu = cos(p);
            // vdW points
            float x = vdW_s * xu + coord[3*ii];
            float y = vdW_s * yu + coord[3*ii+1];
            float z = vdW_s * zu + coord[3*ii+2];
            // Solvent center
            float x2 = r * xu + coord[3*ii];
            float y2 = r * yu + coord[3*ii+1];
            float z2 = r * zu + coord[3*ii+2];
            for (int kk = 0; kk < close_num[ii]; kk++) {
                int atom2i = close_idx[ii * 128 + kk];
                int atom2t = Ele[atom2i];
                float dx = (x - coord[3*atom2i]);
                float dy = (y - coord[3*atom2i+1]);
                float dz = (z - coord[3*atom2i+2]);
                float dr2 = dx * dx + dy * dy + dz * dz; 
                float dx2 = (x2 - coord[3*atom2i]);
                float dy2 = (y2 - coord[3*atom2i+1]);
                float dz2 = (z2 - coord[3*atom2i+2]);
                float dr22 = dx2 * dx2 + dy2 * dy2 + dz2 * dz2;
                // vdW points must not cross into other atom
                if (dr2 < vdW[atom2t] * vdW[atom2t]) pt = 0; //pts[jj] = 0;
                // solvent center has to be far enough
                if (dr22 < (vdW[atom2t]+sol_s) * (vdW[atom2t]+sol_s)) pt = 0; //pts[jj] = 0;
                
            }
            pts[jj] = pt;
        }
        // Sum pts == 1, calc surf area and assign to V[ii]
        for (int stride = num_raster / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                pts[iAccum] += pts[stride + iAccum];
            }
        }
        __syncthreads();
        if (threadIdx.x == 0) {
            V[ii] = (float)pts[0]/(float)num_raster;// * 4.0 * r * r * PI ;
        }
    }
}


__global__ void sum_V (
    float *V, 
    float *V_s, 
    int num_atom, 
    int num_atom2, 
    int *Ele,
    float sol_s, 
    float *vdW) {

    for (int ii = threadIdx.x; ii < num_atom2; ii += blockDim.x) {
        if (ii < num_atom) {
            int atomi = Ele[ii];
            V_s[ii] = V[ii] * 4.0 * PI * (vdW[atomi]+sol_s) * (vdW[atomi]+sol_s);
        } else {
            V_s[ii] = 0.0;
        }
    }
    for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
        __syncthreads();
        for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
            V_s[iAccum] += V_s[stride + iAccum];
        }
    }
    __syncthreads();
    //if (threadIdx.x == 0) printf("Convex contact area = %.3f A^2.\n", V_s[0]);
}

__global__ void FF_calc (
    float *q, 
    float *WK, 
    float *vdW, 
    int num_q, 
    int num_ele, 
    float c1, 
    float r_m, 
    float *FF_table,
    float rho) {

    // Calculate the non-SASA part of form factors per element

    __shared__ float q_pt, q_WK, C1, expC1;
    __shared__ float FF_pt[99]; // num_ele + 1, the last one for water.
    __shared__ float vdW_s[99];
    __shared__ float WK_s[1078]; 
    __shared__ float C1_PI_43_rho;
    if (blockIdx.x >= num_q) return; // out of q range
    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {
        q_pt = q[ii];
        q_WK = q_pt / 4.0 / PI;
        // FoXS C1 term
        expC1 = -powf(4.0 * PI / 3.0, 1.5) * q_WK * q_WK * r_m * r_m * (c1 * c1 - 1.0) / 4.0 / PI;
        C1 = powf(c1,3) * exp(expC1);
        C1_PI_43_rho = C1 * PI * 4.0 / 3.0 * rho;
        for (int jj = threadIdx.x; jj < 11 * num_ele; jj += blockDim.x) {
            WK_s[jj] = WK[jj];
        } // Copy WK to shared memory for faster access
        __syncthreads();

        // Calculate Form factor for this block (or q vector)
        for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
            vdW_s[jj] = vdW[jj];
            if (jj == num_ele) {
                // water
                FF_pt[jj] = WK_s[7*11+5];  // Oxygen
                FF_pt[jj] += 2.0 * WK_s[5];  // Hydrogen
                FF_pt[jj] -= C1_PI_43_rho * powf(vdW_s[jj],3.0) * exp(-PI * powf(4.0/3.0*PI, 2.0/3.0) * vdW_s[jj] * vdW_s[jj] * q_WK * q_WK);  // Water vdW_s
                for (int kk = 0; kk < 5; kk ++) {
                    FF_pt[jj] += WK_s[7*11+kk] * exp(-WK_s[7*11+kk+6] * q_WK * q_WK); // Oxygen
                    FF_pt[jj] += WK_s[kk] * exp(-WK_s[kk+6] * q_WK * q_WK) * 2.0; // Hydrogen
                }
            } else { 
                FF_pt[jj] = WK_s[jj*11+5];
                // The part is for excluded volume
                FF_pt[jj] -= C1_PI_43_rho * powf(vdW_s[jj],3.0) * exp(-PI * powf(4.0/3.0*PI, 2.0/3.0) * vdW_s[jj] * vdW_s[jj] * q_WK * q_WK);  // Water vdW_s
                //FF_pt[jj] -= C1_PI_43_rho * powf(vdW_s[jj],3.0) * exp(-PI * vdW_s[jj] * vdW_s[jj] * q_WK * q_WK);
                for (int kk = 0; kk < 5; kk++) {
                    FF_pt[jj] += WK_s[jj*11+kk] * exp(-WK_s[jj*11+kk+6] * q_WK * q_WK); 
                }
            }
            FF_table[ii*(num_ele+1)+jj] = FF_pt[jj];
        }
    }
}


__global__ void create_FF_full_FoXS (
    float *FF_table, 
    float *V, 
    float c2, 
    int *Ele, 
    float *FF_full, 
    int num_q, 
    int num_ele, 
    int num_atom, 
    int num_atom1024) {

    // Add on SASA for each atom

    __shared__ float FF_pt[99];
    float hydration;
    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        // Get form factor for this block (or q vector)
        if (ii < num_q) {
            for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
                FF_pt[jj] = FF_table[ii*(num_ele+1)+jj];
            }
        }
        __syncthreads();
        
        // In FoXS since c2 remains the same for all elements it is reduced to one value.
        hydration = c2 * FF_pt[num_ele];
        
        // Calculate atomic form factor for this q
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            int atomt = Ele[jj];
            FF_full[ii*num_atom1024 + jj] = FF_pt[atomt] + hydration * V[jj];
        }
    }
}

__global__ void __launch_bounds__(1024,2) scat_calc (
    float *coord, 
    int *Ele,
    float *q,
    float *S_calc, 
    int num_atom,   
    int num_q,     
    int num_ele,   
    float *S_calcc, 
    int num_atom1024,
    float *FF_full) {

    float q_pt;

    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        q_pt = q[ii];

        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            // for every atom jj
            float atom1x = coord[3*jj];
            float atom1y = coord[3*jj+1];
            float atom1z = coord[3*jj+2];
            float S_calccs = 0.0;
            for (int kk = 0; kk < num_atom; kk++) {
                // for every atom kk
                float FF_kj = FF_full[ii * num_atom1024 + jj] * FF_full[ii * num_atom1024 + kk];
                if (q_pt == 0.0 || kk == jj) {
                    S_calccs += FF_kj;
                } else {
                    float dx = atom1x - coord[3*kk];
                    float dy = atom1y - coord[3*kk+1];
                    float dz = atom1z - coord[3*kk+2];
                    float r = sqrt(dx*dx+dy*dy+dz*dz);
                    float qr = q_pt * r; 
                    float sqr = sin(qr) / qr;
                    S_calccs += FF_kj * sqr;
                }
            }
            S_calcc[ii*blockDim.x+threadIdx.x] += S_calccs;
        }
        
        // Tree-like summation of S_calcc to get S_calc
        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * blockDim.x + iAccum] += S_calcc[ii * blockDim.x + stride + iAccum];
            }
        }
        __syncthreads();
        
        S_calc[ii] = S_calcc[ii * blockDim.x];
        __syncthreads();


    }
}


__global__ void __launch_bounds__(1024,2) scat_calc_oa (
    float *coord, 
    int *Ele,
    float *q,
    float *S_calc, 
    int num_atom,   
    int num_q,
    int num_ele,   
    float *S_calcc, 
    int num_atom1024,
    float *FF_full,
    int num_q_raster,
    int num_q_raster2) {

    float q_pt;

    // raster of q points
    // if user set num_q_raster > 1024,
    // it'll get reduced to 1024 before the call
    __shared__ float q_raster[3072];
    __shared__ float L;
    
    L = sqrt(num_q_raster * PI);

    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        q_pt = q[ii];

        for (int jj = threadIdx.x; jj < num_q_raster; jj += blockDim.x) {
            float h = 1.0 - (2.0 * (float)jj + 1.0) / (float)num_q_raster;
            float p = acos(h);
            float t = L * p; 
            float xu = sin(p) * cos(t);
            float yu = sin(p) * sin(t);
            float zu = cos(p);
            // q raster points
            q_raster[3*jj] = q_pt * xu;
            q_raster[3*jj+1] = q_pt * yu;
            q_raster[3*jj+2] = q_pt * zu;
        }
        __syncthreads();
        for (int jj = threadIdx.x; jj < num_q_raster; jj += blockDim.x) {
            // for every atom jj
            float qx = q_raster[3*jj];
            float qy = q_raster[3*jj+1];
            float qz = q_raster[3*jj+2];
            float amp_cos = 0.0; // this q and this q raster point, summed over all atoms
            float amp_sin = 0.0; // this q and this q raster point, summed over all atoms
            for (int kk = 0; kk < num_atom; kk++) {
                float FF = FF_full[ii * num_atom1024 + kk];
                float qrx = -coord[3*kk] * qx;
                float qry = -coord[3*kk+1] * qy;
                float qrz = -coord[3*kk+2] * qz;
                float qr = qrx+qry+qrz;
                amp_cos += FF * cos(qr);
                amp_sin += FF * sin(qr);
            }
            S_calcc[ii*num_q_raster2+jj] = (amp_cos * amp_cos + amp_sin * amp_sin) / float(num_q_raster);
        }
        
        // Tree-like summation of S_calcc to get S_calc
        for (int stride = num_q_raster2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for (int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * num_q_raster2 + iAccum] += S_calcc[ii * num_q_raster2 + stride + iAccum];
            }
        }
        __syncthreads();
        
        S_calc[ii] = S_calcc[ii * num_q_raster2];
        __syncthreads();


    }
}

__global__ void __launch_bounds__(1024,2) scat_calc_oa2 (
    float *coord, 
    int *Ele,
    float *q,
    float *S_calc, 
    int num_atom,   
    int num_q,
    int num_ele,   
    float *S_calcc, 
    int num_atom1024,
    float *FF_full,
    int num_q_raster,
    int num_q_raster2) {

    float q_pt;
    __shared__ float L;
    
    L = sqrt(num_q_raster * PI);

    // raster of q points
    // if user set num_q_raster > 1024,
    // it'll get reduced to 1024 before the call
    __shared__ float q_raster[3072];
    __shared__ float amp_cos[1024];
    __shared__ float amp_sin[1024];
    

    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {
        for (int jj = threadIdx.x; jj < 1024; jj += blockDim.x) {
            amp_cos[jj] = 0.0;
            amp_sin[jj] = 0.0;
        }
        __syncthreads();

        q_pt = q[ii];
        for (int jj = threadIdx.x; jj < num_q_raster; jj += blockDim.x) {
            float h = 1.0 - (2.0 * (float)jj + 1.0) / (float)num_q_raster;
            float p = acos(h);
            float t = L * p; 
            float xu = sin(p) * cos(t);
            float yu = sin(p) * sin(t);
            float zu = cos(p);
            // q raster points
            q_raster[jj*3] = q_pt * xu;
            q_raster[jj*3+1] = q_pt * yu;
            q_raster[jj*3+2] = q_pt * zu;
        }
        __syncthreads(); 

        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            // for every atom jj
            float atomx = coord[3*jj];
            float atomy = coord[3*jj+1];
            float atomz = coord[3*jj+2];
            float FF = FF_full[ii * num_atom1024 + jj];
            for (int kk = 0; kk < num_q_raster; kk++) {
                float qrx = -atomx * q_raster[3*kk];
                float qry = -atomy * q_raster[3*kk+1];
                float qrz = -atomz * q_raster[3*kk+2];
                float qr = qrx+qry+qrz;
                float amp_cos_one = FF * cos(qr);
                float amp_sin_one = FF * sin(qr);
                atomicAdd(&amp_cos[kk], amp_cos_one);
                atomicAdd(&amp_sin[kk], amp_sin_one);
            }
        }

        __syncthreads();
        for (int jj = threadIdx.x; jj < num_q_raster; jj += blockDim.x) {
           S_calcc[ii*num_q_raster2+jj] = (amp_cos[jj] * amp_cos[jj] + amp_sin[jj] * amp_sin[jj]) / float(num_q_raster);
        }    
        // Tree-like summation of S_calcc to get S_calc
        for (int stride = num_q_raster2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * num_q_raster2 + iAccum] += S_calcc[ii * num_q_raster2 + stride + iAccum];
            }
        }
        __syncthreads();
        
        S_calc[ii] = S_calcc[ii * num_q_raster2];
        __syncthreads();


    }
}
