#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <time.h>
#include "WaasKirf.hh"
#define PI 3.14159265359



__global__ void FF_calc (
    float *q, 
    float *WK, 
    int num_q, 
    int num_ele, 
    float *FF_table
    ) {

    // Calculate the non-SASA part of form factors per element

    __shared__ float q_pt, q_WK;
    __shared__ float FF_pt[98]; // num_ele + 1, the last one for water.
    __shared__ float WK_s[1078]; 
    if (blockIdx.x >= num_q) return; // out of q range
    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {
        q_pt = q[ii];
        q_WK = q_pt / 4.0 / PI;
        // FoXS C1 term
        for (int jj = threadIdx.x; jj < 11 * num_ele; jj += blockDim.x) {
            WK_s[jj] = WK[jj];
        } // Copy WK to shared memory for faster access
        __syncthreads();

        // Calculate Form factor for this block (or q vector)
        for (int jj = threadIdx.x; jj < num_ele; jj += blockDim.x) {
            FF_pt[jj] = WK_s[jj*11+5];
            // The part is for excluded volume
            //FF_pt[jj] -= C1_PI_43_rho * powf(vdW_s[jj],3.0) * exp(-PI * vdW_s[jj] * vdW_s[jj] * q_WK * q_WK);
            for (int kk = 0; kk < 5; kk++) {
                FF_pt[jj] += WK_s[jj*11+kk] * exp(-WK_s[jj*11+kk+6] * q_WK * q_WK); 
            }
            FF_table[ii*(num_ele)+jj] = FF_pt[jj];
        }
    }
}


__global__ void create_FF_full_FoXS (
    float *FF_table, 
    int *Ele, 
    float *FF_full, 
    int num_q, 
    int num_ele, 
    int num_atom, 
    int num_atom1024) {

    // Add on SASA for each atom

    __shared__ float FF_pt[99];
    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        // Get form factor for this block (or q vector)
        if (ii < num_q) {
            for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
                FF_pt[jj] = FF_table[ii*num_ele+jj];
            }
        }
        __syncthreads();
        
        // Calculate atomic form factor for this q
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            int atomt = Ele[jj];
            FF_full[ii*num_atom1024 + jj] = FF_pt[atomt];
        }
    }
}

__global__ void __launch_bounds__(1024,2) scat_calc (
    float *coord, 
    int *Ele,
    float *q,
    float *S_calc, 
    int num_atom,   
    int num_q,     
    int num_ele,   
    float *S_calcc, 
    int num_atom1024,
    float *FF_full) {

    float q_pt;

    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        q_pt = q[ii];

        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            // for every atom jj
            float atom1x = coord[3*jj];
            float atom1y = coord[3*jj+1];
            float atom1z = coord[3*jj+2];
            float S_calccs = 0.0;
            for (int kk = 0; kk < num_atom; kk++) {
                // for every atom kk
                float FF_kj = FF_full[ii * num_atom1024 + jj] * FF_full[ii * num_atom1024 + kk];
                if (q_pt == 0.0 || kk == jj) {
                    S_calccs += FF_kj;
                } else {
                    float dx = atom1x - coord[3*kk];
                    float dy = atom1y - coord[3*kk+1];
                    float dz = atom1z - coord[3*kk+2];
                    float r = sqrt(dx*dx+dy*dy+dz*dz);
                    float qr = q_pt * r; 
                    float sqr = sin(qr) / qr;
                    S_calccs += FF_kj * sqr;
                }
            }
            S_calcc[ii*blockDim.x+threadIdx.x] += S_calccs;
        }
        
        // Tree-like summation of S_calcc to get S_calc
        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * blockDim.x + iAccum] += S_calcc[ii * blockDim.x + stride + iAccum];
            }
        }
        __syncthreads();
        
        S_calc[ii] = S_calcc[ii * blockDim.x];
        __syncthreads();


    }
}


__global__ void __launch_bounds__(1024,2) scat_calc_oa (
    float *coord, 
    int *Ele,
    float *q,
    float *S_calc, 
    int num_atom,   
    int num_q,
    int num_ele,   
    float *S_calcc, 
    int num_atom1024,
    float *FF_full,
    int num_q_raster,
    int num_q_raster2) {

    float q_pt;

    // raster of q points
    // if user set num_q_raster > 1024,
    // it'll get reduced to 1024 before the call

    float L = sqrt(num_q_raster * PI);

    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        q_pt = q[ii];

        for (int jj = threadIdx.x; jj < num_q_raster; jj += blockDim.x) {
            float h = 1.0 - (2.0 * (float)jj + 1.0) / (float)num_q_raster;
            float p = acos(h);
            float t = L * p; 
            float xu = sin(p) * cos(t);
            float yu = sin(p) * sin(t);
            float zu = cos(p);
            // q raster points
            float qx = q_pt * xu;
            float qy = q_pt * yu;
            float qz = q_pt * zu;
            float amp_cos = 0.0; // this q and this q raster point, summed over all atoms
            float amp_sin = 0.0; // this q and this q raster point, summed over all atoms
            for (int kk = 0; kk < num_atom; kk++) {
                float FF = FF_full[ii * num_atom1024 + kk];
                float qrx = -coord[3*kk] * qx;
                float qry = -coord[3*kk+1] * qy;
                float qrz = -coord[3*kk+2] * qz;
                float qr = qrx + qry + qrz;
                amp_cos += FF * cos(qr);
                amp_sin += FF * sin(qr);
            }
            S_calcc[ii*num_q_raster2+jj] = (amp_cos * amp_cos + amp_sin * amp_sin) / float(num_q_raster);
        }
        
        // Tree-like summation of S_calcc to get S_calc
        for (int stride = num_q_raster2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for (int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * num_q_raster2 + iAccum] += S_calcc[ii * num_q_raster2 + stride + iAccum];
            }
        }
        __syncthreads();
        
        S_calc[ii] = S_calcc[ii * num_q_raster2];
        __syncthreads();


    }
}

__global__ void __launch_bounds__(1024,2) scat_calc_oa2 (
    float *coord, 
    int *Ele,
    float *q,
    float *S_calc, 
    int num_atom,   
    int num_q,
    int num_ele,   
    float *S_calcc, 
    int num_atom1024,
    float *FF_full,
    int num_q_raster,
    int num_q_raster2) {

    float q_pt;
    __shared__ float L;
    
    L = sqrt(num_q_raster * PI);

    // raster of q points
    // if user set num_q_raster > 1024,
    // it'll get reduced to 1024 before the call
    __shared__ float q_raster[3072];
    __shared__ float amp_cos[1024];
    __shared__ float amp_sin[1024];
    

    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {
        for (int jj = threadIdx.x; jj < 1024; jj += blockDim.x) {
            amp_cos[jj] = 0.0;
            amp_sin[jj] = 0.0;
        }
        __syncthreads();

        q_pt = q[ii];
        for (int jj = threadIdx.x; jj < num_q_raster; jj += blockDim.x) {
            float h = 1.0 - (2.0 * (float)jj + 1.0) / (float)num_q_raster;
            float p = acos(h);
            float t = L * p; 
            float xu = sin(p) * cos(t);
            float yu = sin(p) * sin(t);
            float zu = cos(p);
            // q raster points
            q_raster[jj*3] = q_pt * xu;
            q_raster[jj*3+1] = q_pt * yu;
            q_raster[jj*3+2] = q_pt * zu;
        }
        __syncthreads(); 

        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            // for every atom jj
            float atomx = coord[3*jj];
            float atomy = coord[3*jj+1];
            float atomz = coord[3*jj+2];
            float FF = FF_full[ii * num_atom1024 + jj];
            for (int kk = 0; kk < num_q_raster; kk++) {
                float qrx = -atomx * q_raster[3*kk];
                float qry = -atomy * q_raster[3*kk+1];
                float qrz = -atomz * q_raster[3*kk+2];
                float qr = qrx+qry+qrz;
                float amp_cos_one = FF * cos(qr);
                float amp_sin_one = FF * sin(qr);
                atomicAdd(&amp_cos[kk], amp_cos_one);
                atomicAdd(&amp_sin[kk], amp_sin_one);
            }
        }

        __syncthreads();
        for (int jj = threadIdx.x; jj < num_q_raster; jj += blockDim.x) {
           S_calcc[ii*num_q_raster2+jj] = (amp_cos[jj] * amp_cos[jj] + amp_sin[jj] * amp_sin[jj]) / float(num_q_raster);
        }    
        // Tree-like summation of S_calcc to get S_calc
        for (int stride = num_q_raster2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * num_q_raster2 + iAccum] += S_calcc[ii * num_q_raster2 + stride + iAccum];
            }
        }
        __syncthreads();
        
        S_calc[ii] = S_calcc[ii * num_q_raster2];
        __syncthreads();


    }
}
